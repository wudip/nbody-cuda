#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "simple_cell.h"

#define SOFTENING_FACTOR_SQR 0.5
#define GRAVITATION_CONSTANT 6.67300E-11

#define IS_NOT_EMPTY(index) ((unsigned int) -1) != index

SimpleCell::SimpleCell() {}

SimpleCell::SimpleCell(unsigned int offset, unsigned int particle, const Particle &center, unsigned int parent,
                       unsigned int *sub) :
        offset(offset), particle(particle), center(center), parent(parent) {
    for (int i = 0; i < NUM_OF_SUBCELLS; ++i) {
        subtree[i] = sub[i];
    }
}

__device__ const SimpleCell *SimpleCell::getCell(unsigned int position) const {
    int diff = position - offset;
    return this + diff;
}

__device__ void SimpleCell::getForceSiblings(const Particle &refParticle, Vec3<double> &forces) const {
    for (int i = 0; i < NUM_OF_SUBCELLS; ++i) {
        unsigned int sibIndex = getCell(parent)->subtree[i];
        if (sibIndex == offset) continue;
        const SimpleCell *sibling = getCell(sibIndex);
        const Particle &sibPart = sibling->center;
        addToForces(forces, refParticle, sibPart);
    }
}

__device__ Vec3<double> SimpleCell::getForce(const Particle *particles) const {
    Vec3<double> force(0, 0, 0);
    const SimpleCell *c = this;
    while (IS_NOT_EMPTY(c->parent)) {
        getForceSiblings(particles[c->particle], force);
        c = getCell(c->parent);
    }
    return force;
}

__device__ void addToForces(Vec3<double> &forces, const Particle &particle, const Particle &sibPart) {
    Vec3<double> diff = particle.getPosition() - sibPart.getPosition();
    double bottom = pow(diff.sqrSize() + SOFTENING_FACTOR_SQR, 1.5);
    double massTotal = GRAVITATION_CONSTANT * particle.mass * sibPart.mass;
    forces += diff * massTotal / bottom;
}
