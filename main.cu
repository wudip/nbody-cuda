#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>
#include <fstream>
#include <utility>

#include "main.h"
#include "cell.h"

// Softening factor squared
#define SOFTENING_FACTOR_SQR 0.5
#define GRAVITATION_CONSTANT 6.67300E-11
#define WINDOWS_SUCKS 1
#define NUM_REPEATS 5

using namespace std;

int main(int argc, char **argv) {
    // Load input
    pair<Particle*, unsigned int> input;
    if (WINDOWS_SUCKS && argc > 1) {
        std::ifstream ifs;
        ifs.open(argv[1], ifstream::in);
        input = loadParticles(ifs);
        ifs.close();
    } else {
        input = loadParticles(cin);
    }
    particles = input.first;
    unsigned int size = input.second;
    if (particles == null) return;

    // Actual algorithm
    clock_t clk_start = clock();
    for (int i = 0; i < NUM_REPEATS; ++i) {
        // Create octree
        double *particleBoundaries = computeParticleBoundaries(particles, size);
        Cell octree(particleBoundaries, particleBoundaries + 3);
        delete[] particleBoundaries;
        for (unsigned int partIndex = 0; partIndex < size; ++partIndex) {
            octree.add(particles + partIndex);
        }
        octree.updateCenter();
        // Solve N-body problem
        nbodyBarnesHut(particles, size, octree);
    }
    clock_t clk_end = clock();
    printParticles(particles, size, cout);
    delete[] particles;
    long time = (clk_end - clk_start) / (CLOCKS_PER_SEC / 1000);
    cout << "Time: " << time << " ms" << endl;
    return 0;
}

pair<Particle *, unsigned int> loadParticles(istream &input) {
    // Loading vector
    vector<Particle> particles;
    double x;
    double y;
    double z;
    double mass;
    while (input >> x && input >> y && input >> z && input >> mass) {
        particles.push_back(Particle(x, y, z, mass));
    }

    // Move particles from vector to array
    unsigned int size = (unsigned int) particles->size();
    Particle* particleArr = size == 0 ? null : new Particle[size];
    for (auto pit = particles->begin(); pit < particles->end(); ++pit) {
        particleArr[pit - particles->begin()] = * pit;
    }

    return pair(particleArr, size);
}

vector<Vec3<double>> nbody(const vector<Particle> *particles) {
    vector<Vec3<double>> forces;
    forces.reserve(particles->size());
    for (auto it1 = particles->begin(); it1 < particles->end(); ++it1) {
        Vec3<double> res(0, 0, 0);
        for (auto it2 = particles->begin(); it2 < particles->end(); ++it2) {
            if (it1 == it2) continue;
            Vec3<double> diff = it1->getPosition() - it2->getPosition();
            double bottom = pow(diff.sqrSize() + SOFTENING_FACTOR_SQR, 1.5);
            double massTotal = GRAVITATION_CONSTANT * it1->mass * it2->mass;
            res += diff * massTotal / bottom;
        }
        forces.push_back(res);
    }
    return forces;
}

__global__ void nbodyBarnesHutCuda(
        SimpleCell * cells,
        unsigned int * partPositions,
        Particle * particles,
        Vec3<double>* forces,
        unsigned int nOfParticles,
        unsigned int offset)
{
    unsigned int index = offset + threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= nOfParticles) return;
    SimpleCell *particleCell = cells + partPositions[index];
    Vec3<double> force = particleCell->getForce(particles) * 10000;
    Vec3<double> acceleration = force / particles[index].mass;
//    printf("Acc: %lf\n", acceleration);
    forces[index] = acceleration;
    //particles[index].accelerate(acceleration);
    //particles[index].updatePosition();
}

void nbodyBarnesHut(Particle *particles, unsigned int nOfParticles, Cell &cell) {
    unsigned int nOfCells;
    pair<SimpleCell *, unsigned int *> serialized = cell.serialize(particles, nOfCells);
    SimpleCell *flatTree = serialized.first;
    unsigned int *partPositions = serialized.second;

    SimpleCell *flatTreeCuda;
    unsigned int * partPositionsCuda;
    Particle * particlesCuda;
    Vec3<double>* forcesCuda;

    hipMalloc((void**)&flatTreeCuda, (nOfCells) * sizeof(SimpleCell));
    hipMalloc((void**)&partPositionsCuda, (nOfParticles) * sizeof(unsigned int));
    hipMalloc((void**)&particlesCuda, (nOfParticles) * sizeof(Particle));
    hipMalloc((void**)&forcesCuda, (nOfParticles) * sizeof(Vec3<double>));

    hipMemcpy(flatTreeCuda, flatTree, sizeof(SimpleCell)*(nOfCells), hipMemcpyHostToDevice);
    hipMemcpy(partPositionsCuda, partPositions, sizeof(unsigned int)*(nOfParticles), hipMemcpyHostToDevice);
    hipMemcpy(particlesCuda, particles, sizeof(Particle)*(nOfParticles), hipMemcpyHostToDevice);

    int nOfBlocks = (nOfParticles - 1) / 1024 + 1;
    nbodyBarnesHutCuda <<<nOfBlocks,  min(nOfParticles, 1024)>>>(flatTreeCuda, partPositionsCuda, particlesCuda, forcesCuda, nOfParticles, 0);

    hipDeviceSynchronize();

    moveParticles<<<1,1>>>(particlesCuda, forcesCuda, nOfParticles);

    hipDeviceSynchronize();

    hipMemcpy(particles, particlesCuda, sizeof(Particle)*(nOfParticles), hipMemcpyDeviceToHost);

    hipFree(flatTreeCuda);
    hipFree(partPositionsCuda);
    hipFree(particlesCuda);
    hipFree(forcesCuda);

    delete[] flatTree;
    delete[] partPositions;
}

__global__ void moveParticles(Particle *particles, const Vec3<double> *forces, unsigned int n) {
    auto particleIt = particles;
    auto forceIt = forces;
    for(unsigned int i = 0; i < n; i++) {
        Vec3<double> acceleration = *forceIt / particleIt->mass;
        particleIt->accelerate(acceleration);
        particleIt->updatePosition();
        ++particleIt;
        ++forceIt;
    }
}

/**
 * Computes coordinates of smallest possible box containing all the particles
 * @return array of minimum point and maximum point: [min_x, min_y, min_z, max_x, max_y, max_z]
 */
double *computeParticleBoundaries(const Particle* particles, unsigned int size) {
    double *result = new double[6];
    for (int i = 0; i < 3; ++i) {
        result[i] = INFINITY;
        result[3 + i] = -INFINITY;
    }
    for (const Particle* it = particles; it < particles + size; ++it) {
        Vec3<double> pos = it->getPosition();
        for (int dim = 0; dim < 3; ++dim) {
            double coord = pos.getDim(dim);
            if (coord < result[dim]) {
                result[dim] = coord;
            }
            if (coord > result[3 + dim]) {
                result[3 + dim] = coord;
            }
        }
    }
    return result;
}

void printParticles(const Particle *particles, unsigned int size, ostream &out) {
    for (unsigned int i = 0; i < size; ++i) {
        out << particles[i] << endl;
    }
}

