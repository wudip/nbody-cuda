#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>
#include <fstream>

#include "main.h"
#include "cell.h"

// Softening factor squared
#define SOFTENING_FACTOR_SQR 0.5
#define GRAVITATION_CONSTANT 6.67300E-11
#define WINDOWS_SUCKS 1

using namespace std;

int main(int argc, char **argv) {
    vector<Particle> *particles;
    if (WINDOWS_SUCKS && argc > 1) {
        std::ifstream ifs;
        ifs.open(argv[1], ifstream::in);
        particles = loadParticles(ifs);
        ifs.close();
    } else {
        particles = loadParticles(cin);
    }
    clock_t clk_start = clock();
    for (int i = 0; i < 1; ++i) {
        // Create octree
        double *particleBoundaries = computeParticleBoundaries(particles);

        // move particles from vector to array
        unsigned int size = (unsigned int) particles->size();
        Particle *particleArr = new Particle[size];
        for (auto pit = particles->begin(); pit < particles->end(); ++pit) {
            particleArr[pit - particles->begin()] = *pit;
        }

        Cell octree(particleBoundaries, particleBoundaries + 3);
        delete[] particleBoundaries;
        for (unsigned int partIndex = 0; partIndex < size; ++partIndex) {
            octree.add(particleArr + partIndex);
        }
        octree.updateCenter();


        nbodyBarnesHut(particleArr, size, octree);

        // delete new particles
        for (auto pit = particles->begin(); pit < particles->end(); ++pit) {
            *pit = particleArr[pit - particles->begin()];
        }
        delete[] particleArr;

        //vector<Vec3<double>> forces = nbody(particles);
    }
    clock_t clk_end = clock();
    cout << "Time: " << (clk_end - clk_start) << " ms" << endl;
    printParticles(particles, cout);
    delete particles;
    return 0;
}

vector<Particle> *loadParticles(istream &input) {
    vector<Particle> *particles = new vector<Particle>();
    double x;
    double y;
    double z;
    double mass;
    while (input >> x && input >> y && input >> z && input >> mass) {
        particles->push_back(Particle(x, y, z, mass));
    }
    return particles;
}

vector<Vec3<double>> nbody(const vector<Particle> *particles) {
    vector<Vec3<double>> forces;
    forces.reserve(particles->size());
    for (auto it1 = particles->begin(); it1 < particles->end(); ++it1) {
        Vec3<double> res(0, 0, 0);
        for (auto it2 = particles->begin(); it2 < particles->end(); ++it2) {
            if (it1 == it2) continue;
            Vec3<double> diff = it1->getPosition() - it2->getPosition();
            double bottom = pow(diff.sqrSize() + SOFTENING_FACTOR_SQR, 1.5);
            double massTotal = GRAVITATION_CONSTANT * it1->mass * it2->mass;
            res += diff * massTotal / bottom;
        }
        forces.push_back(res);
    }
    return forces;
}

__global__ void nbodyBarnesHutCuda(
        SimpleCell * cells,
        unsigned int * partPositions,
        Particle * particles,
        Vec3<double>* forces,
        unsigned int nOfParticles,
        unsigned int offset)
{
    unsigned int index = offset + threadIdx.x + blockIdx.x * blockDim.x;
    printf("Jezinka\n");
    if (index >= nOfParticles) return;
    printf("Jelen c. %d\n", partPositions[index]);
    SimpleCell *particleCell = cells + partPositions[index];
    printf("Smolicek pacholicek\n");
    Vec3<double> force = particleCell->getForce(particles);
    printf("Force: %lf %lf %lf\n", force.x, force.y, force.z);
    Vec3<double> acceleration = force / particles[index].mass;
//    printf("Acc: %lf\n", acceleration);
    forces[index] = acceleration;
    //particles[index].accelerate(acceleration);
    //particles[index].updatePosition();
}

void nbodyBarnesHut(Particle *particles, unsigned int nOfParticles, Cell &cell) {
    unsigned int nOfCells;
    pair<SimpleCell *, unsigned int *> serialized = cell.serialize(particles, nOfCells);
    SimpleCell *flatTree = serialized.first;
    unsigned int *partPositions = serialized.second;

    SimpleCell *flatTreeCuda;
    unsigned int * partPositionsCuda;
    Particle * particlesCuda;
    Vec3<double>* forcesCuda;

    hipMalloc((void**)&flatTreeCuda, (nOfCells) * sizeof(SimpleCell));
    hipMalloc((void**)&partPositionsCuda, (nOfParticles) * sizeof(unsigned int));
    hipMalloc((void**)&particlesCuda, (nOfParticles) * sizeof(Particle));
    hipMalloc((void**)&forcesCuda, (nOfParticles) * sizeof(Vec3<double>));

    hipMemcpy(flatTreeCuda, flatTree, sizeof(SimpleCell)*(nOfCells), hipMemcpyHostToDevice);
    hipMemcpy(partPositionsCuda, partPositions, sizeof(unsigned int)*(nOfParticles), hipMemcpyHostToDevice);
    hipMemcpy(particlesCuda, particles, sizeof(Particle)*(nOfParticles), hipMemcpyHostToDevice);


    nbodyBarnesHutCuda <<<1,  nOfParticles>>>(flatTreeCuda, partPositionsCuda, particlesCuda, forcesCuda, nOfParticles, 0);

    hipDeviceSynchronize();

    moveParticles<<<1,1>>>(particlesCuda, forcesCuda, nOfParticles);
    
    hipDeviceSynchronize();

    hipMemcpy(particles, particlesCuda, sizeof(Particle)*(nOfParticles), hipMemcpyDeviceToHost);

    hipFree(flatTreeCuda);
    hipFree(partPositionsCuda);
    hipFree(particlesCuda);
    hipFree(forcesCuda);

    delete[] flatTree;
    delete[] partPositions;
}

__global__ void moveParticles(Particle *particles, const Vec3<double> *forces, unsigned int n) {
    auto particleIt = particles;
    auto forceIt = forces;
    for(unsigned int i = 0; i < n; i++) {
        Vec3<double> acceleration = *forceIt / particleIt->mass;
        particleIt->accelerate(acceleration);
        particleIt->updatePosition();
        ++particleIt;
        ++forceIt;
    }
}

/**
 * Computes coordinates of smallest possible box containing all the particles
 * @return array of minimum point and maximum point: [min_x, min_y, min_z, max_x, max_y, max_z]
 */
double *computeParticleBoundaries(const vector<Particle> *particles) {
    double *result = new double[6];
    for (int i = 0; i < 3; ++i) {
        result[i] = INFINITY;
        result[3 + i] = -INFINITY;
    }
    for (auto it = particles->begin(); it < particles->end(); ++it) {
        Vec3<double> pos = it->getPosition();
        for (int dim = 0; dim < 3; ++dim) {
            double coord = pos.getDim(dim);
            if (coord < result[dim]) {
                result[dim] = coord;
            }
            if (coord > result[3 + dim]) {
                result[3 + dim] = coord;
            }
        }
    }
    return result;
}

void printParticles(const vector<Particle> *particles, ostream &out) {
    for (auto it = particles->begin(); it < particles->end(); ++it) {
        cout << *it << endl;
    }
}
