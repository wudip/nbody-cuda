#include "hip/hip_runtime.h"

#include <cmath>
#include "vec3.h"

template<typename T>
__host__ __device__ Vec3<T>::Vec3() : x(0), y(0), z(0) {};

template<typename T>
__host__ __device__ Vec3<T>::Vec3(T x, T y, T z) {
    this->x = x;
    this->y = y;
    this->z = z;
}

template<typename T>
void Vec3<T>::set(const T &x, const T &y, const T &z) {
    this->x = x;
    this->y = y;
    this->z = z;
}

template<typename T>
void Vec3<T>::normalise() {
    T magnitude = sqrt((x * x) + (y * y) + (z * z));
    if (magnitude != 0) {
        x /= magnitude;
        y /= magnitude;
        z /= magnitude;
    }
}

template<typename T>
void Vec3<T>::square() {
    x *= x;
    y *= y;
    z *= z;
}

template<typename T>
T Vec3<T>::dotProduct(const Vec3<T> &a, const Vec3<T> &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

template<typename T>
T Vec3<T>::dotProduct(const Vec3<T> &vec) const {
    return x * vec.x + y * vec.y + z * vec.z;
}

template<typename T>
__host__ __device__ T Vec3<T>::getDistance(const Vec3<T> &a, const Vec3<T> &b) {
    T dx = b.x - a.x;
    T dy = b.y - a.y;
    T dz = b.z - a.z;
    return sqrt(dx * dx + dy * dy + dz * dz);
}


template<typename T>
__host__ __device__ Vec3<T> Vec3<T>::operator+(const Vec3<T> &vector) const {
    return Vec3<T>(x + vector.x, y + vector.y, z + vector.z);
}

template<typename T>
__host__ __device__ void Vec3<T>::operator+=(const Vec3<T> &vector) {
    x += vector.x;
    y += vector.y;
    z += vector.z;
}

template<typename T>
__host__ __device__ Vec3<T> Vec3<T>::operator-(const Vec3<T> &vector) const {
    return Vec3<T>(x - vector.x, y - vector.y, z - vector.z);
}

template<typename T>
__host__ __device__ void Vec3<T>::operator-=(const Vec3<T> &vector) {
    x -= vector.x;
    y -= vector.y;
    z -= vector.z;
}

template<typename T>
__host__ __device__ Vec3<T> Vec3<T>::operator*(const Vec3<T> &vector) const {
    return Vec3<T>(x * vector.x, y * vector.y, z * vector.z);
}

template<typename T>
__host__ __device__ Vec3<T> Vec3<T>::operator*(const T &value) const {
    return Vec3<T>(x * value, y * value, z * value);
}

template<typename T>
__host__ __device__ void Vec3<T>::operator*=(const T &value) {
    x *= value;
    y *= value;
    z *= value;
}

// template<typename T>
// Vec3<T> Vec3<T>::operator/(const T &value) const {
//     return Vec3<T>(x / value, y / value, z / value);
// }

template<typename T>
__host__ __device__ Vec3<T> Vec3<T>::operator/(const T &value) const {
    return Vec3<T>(x / value, y / value, z / value);
}

template<typename T>
__host__ __device__ void Vec3<T>::operator/=(const T &value) {
    x /= value;
    y /= value;
    z /= value;
}

template<typename T>
__host__ __device__ T Vec3<T>::sqrSize() {
    return x * x + y * y + z * z;
}

template<typename T>
double Vec3<T>::getDim(int dimension) const {
    if (dimension == 0) return x;
    if (dimension == 1) return y;
    if (dimension == 2) return z;
    throw "Dimension out of range";
}

template<typename T>
void Vec3<T>::setDim(int dimension, const double value) {
    if (dimension == 0) x = value;
    else if (dimension == 1) y = value;
    else if (dimension == 2) z = value;
    else throw "Dimension out of range";
}

template
class Vec3<double>;
